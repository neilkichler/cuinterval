#include "hip/hip_runtime.h"
#include <vector>

#include <cuinterval/cuinterval.h>

// #include <thrust/execution_policy.h>
// #include <thrust/fill.h>
// #include <thrust/functional.h>
// #include <thrust/sequence.h>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform.h>

using cu::interval;

struct to_interval_fn
{
    template<typename T>
    __host__ __device__ interval<T> operator()(const T &x) const
    {
        return interval<T> { x, x };
    }
};

struct pi_recip_fn
{
    template<typename I>
    __device__ I operator()(const I &x) const
    {
        return recip(sqr(x));
    }
};

struct pi_pow_fn
{
    template<typename I>
    __device__ I operator()(const I &x) const
    {
        return recip(pow(x, 2));
    }
};

struct pi_inv_fn
{
    template<typename I>
    __device__ I operator()(const I &x) const
    {
        return I { 1.0, 1.0 } / (sqr(x));
    }
};

struct scale_fn
{
    template<typename I>
    __device__ I operator()(I x) const
    {
        return sqrt(x * I { 6.0, 6.0 });
    }
};

struct final_decrement_fn
{
    int n;

    final_decrement_fn(int _n)
        : n(_n)
    { }

    template<typename I>
    __device__ I operator()(I x) const
    {
        I n_lb = I { 0.0 + n, 0.0 + n };
        I n_ub = I { 1.0 + n, 1.0 + n };

        I inv_lb = recip(n_lb);
        I inv_ub = recip(n_ub);

        return x + I { inv_ub.lb, inv_lb.ub };
    }
};

std::vector<interval<double>> compute_pi_approximation(hipStream_t stream)
{
    using T = double;
    using I = interval<T>;

    constexpr int n = 100'000;
    thrust::counting_iterator<T> seq_first(1);
    thrust::counting_iterator<T> seq_last = seq_first + n;

    auto tr_first     = thrust::make_transform_iterator(seq_first, to_interval_fn());
    auto tr_last      = thrust::make_transform_iterator(seq_last, to_interval_fn());
    auto pi_rcp_first = thrust::make_transform_iterator(tr_first, pi_recip_fn());
    auto pi_rcp_last  = thrust::make_transform_iterator(tr_last, pi_recip_fn());
    auto pi_inv_first = thrust::make_transform_iterator(tr_first, pi_inv_fn());
    auto pi_inv_last  = thrust::make_transform_iterator(tr_last, pi_inv_fn());

    I sum_rcp = thrust::reduce(thrust::cuda::par.on(stream), pi_rcp_first, pi_rcp_last, I {});
    I sum_pow = thrust::reduce(thrust::cuda::par.on(stream), pi_rcp_first, pi_rcp_last, I {});
    I sum_inv = thrust::reduce(thrust::cuda::par.on(stream), pi_inv_first, pi_inv_last, I {});

    // NOTE: The rest could (and normally should) be done on the CPU
    //       but for testing purposes we use the GPU.
    thrust::device_vector<I> d_pi { sum_rcp, sum_pow, sum_inv };

    thrust::transform(thrust::cuda::par.on(stream), d_pi.begin(), d_pi.end(), d_pi.begin(), final_decrement_fn(n));
    thrust::transform(thrust::cuda::par.on(stream), d_pi.begin(), d_pi.end(), d_pi.begin(), scale_fn());

    std::vector<I> h_pi(d_pi.size());
    thrust::copy(d_pi.begin(), d_pi.end(), h_pi.begin());

    return h_pi;
}

struct coeff_fn
{
    template<typename T>
    __device__ interval<T> operator()(T x) const
    {
        using I = interval<T>;
        return I { 1.0, 1.0 } / I { x, x };
    }
};

template<typename I>
struct horner_fn
{
    I x;

    horner_fn(I _x)
        : x(_x)
    { }

    __device__ I operator()(I res, I coeff) const
    {
        return res * x + coeff;
    }
};

std::vector<interval<double>> compute_horner(hipStream_t stream)
{
    using T = double;
    using I = interval<T>;

    // Approximate exp with Horner's scheme.
    constexpr int n_coefficients = 16;
    thrust::host_vector<T> ps(n_coefficients);

    thrust::counting_iterator<T> seq_first(1);
    thrust::counting_iterator<T> seq_last = seq_first + n_coefficients;

    thrust::inclusive_scan(seq_first, seq_last, ps.begin(), thrust::multiplies<T>());

    thrust::device_vector<T> d_ps = ps;
    thrust::device_vector<I> d_coefficients(n_coefficients);
    thrust::transform(d_ps.begin(), d_ps.end() - 1, d_coefficients.begin() + 1, coeff_fn());
    d_coefficients[0] = I { 1.0, 1.0 };

    // example input
    T eps = 1.0e-12;
    I x { 1.0 - eps, 1.0 + eps };
    thrust::device_vector<I> d_res(n_coefficients);
    thrust::inclusive_scan(d_coefficients.rbegin(), d_coefficients.rend(), d_res.begin(), horner_fn<I>(x));

    thrust::host_vector<I> coefficients = d_coefficients;

    std::vector<I> h_res(d_res.size());
    thrust::copy(d_res.begin(), d_res.end(), h_res.begin());

    return h_res;
}
