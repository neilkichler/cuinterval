#include "hip/hip_runtime.h"

#include <cuinterval/cuinterval.h>

#include <stdio.h>
#include <stdlib.h>

// compiler bug fix; TODO: remove when fixed
#ifdef __HIPCC__
#pragma push_macro("__cpp_consteval")
#define consteval constexpr
#include <boost/ut.hpp>
#undef consteval
#pragma pop_macro("__cpp_consteval")
#else
#include <boost/ut.hpp>
#endif

#define CUDA_CHECK(x)                                                                \
    do {                                                                             \
        hipError_t err = x;                                                         \
        if (err != hipSuccess) {                                                    \
            fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, \
                    __FILE__, __LINE__, hipGetErrorString(err),                     \
                    hipGetErrorName(err), err);                                     \
            abort();                                                                 \
        }                                                                            \
    } while (0)

template<typename T>
__global__ void test_neg(int n, interval<T> *x)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        x[i] = -x[i];
    }
}

template<typename T>
__global__ void test_add(int n, interval<T> *x, interval<T> *y)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        x[i] = x[i] + y[i];
    }
}

template<typename T>
__global__ void test_sub(int n, interval<T> *x, interval<T> *y)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        x[i] = x[i] - y[i];
    }
}

int main()
{
    using namespace boost::ut;

    using I = interval<double>;

    suite arith_tests = [] {
        I empty = ::empty<double>();
        // I entire = ::entire<double>();

        const int n = 8;
        int n_bytes = n * sizeof(I);

        interval<double> *d_vec;
        CUDA_CHECK(hipMalloc(&d_vec, n_bytes));

        "neg"_test = [&] {
            I h_xs[n] = {
                { 0, 1 },
                { 1, 2 },
                // empty,
                { 0, 2 },
                { -0, 2 },
                { -2, 0 },
                { -2, -0 },
                { 0, 0 },
                { -0, 0 },
            };

            I h_ref[n] = {
                { -1, -0 },
                { -2, -1 },
                // empty,
                { -2, 0 },
                { -2, 0 },
                { 0, 2 },
                { 0, 2 },
                { 0, 0 },
                { 0, 0 },
            };

            CUDA_CHECK(hipMemcpy(d_vec, h_xs, n_bytes, hipMemcpyHostToDevice));

            int blockSize = 256;
            int numBlocks = (n + blockSize - 1) / blockSize;

            test_neg<<<numBlocks, blockSize>>>(n, d_vec);

            CUDA_CHECK(hipMemcpy(h_xs, d_vec, n_bytes, hipMemcpyDeviceToHost));

            for (int i = 0; i < n; ++i) {
                expect(h_xs[i] == h_ref[i]);
                // printf("h_in: %lf, h_ref: %lf\n", h_xs[i].lb, h_ref[i].lb);
                // printf("h_in: %lf, h_ref: %lf\n", h_xs[i].ub, h_ref[i].ub);
            }
        };

        // "add"_test = [&] {
        //     for (int i = 0; i < n; ++i) {
        //         expect(h_xs[i] == h_ref[i]);
        //     }
        // };

        CUDA_CHECK(hipFree(d_vec));
    };

    return 0;
}
