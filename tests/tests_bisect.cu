#include "hip/hip_runtime.h"
#include <cuinterval/cuinterval.h>

#include "tests.h"
#include "test_ops.cuh"

template<typename T>
void tests_bisect() {
    using namespace boost::ut;

    using I = interval<T>;
    using B = bool;
    using N = int;

    T infinity = std::numeric_limits<T>::infinity();
    I empty    = { infinity, -infinity };
    I entire   = { -infinity, infinity };
    T NaN = ::nan("");

    const int n = 8; // count of largest test array
    const int n_bytes   = n * sizeof(I);
    const int blockSize = 256;
    [[maybe_unused]] const int numBlocks = (n + blockSize - 1) / blockSize;

    I *d_xs_, *d_ys_, *d_zs_, *d_res_;

    CUDA_CHECK(hipMalloc(&d_xs_, n_bytes));
    CUDA_CHECK(hipMalloc(&d_ys_, n_bytes));
    CUDA_CHECK(hipMalloc(&d_zs_, n_bytes));
    CUDA_CHECK(hipMalloc(&d_res_, 2*n_bytes));

    "bisection"_test = [&] {
        constexpr int n = 8;
        std::array<I, n> h_xs {{
            empty,
            entire,
            entire,
            entire,
            {0.0, 2.0},
            {1.0, 1.0},
            {0.0, 1.0},
            {0.0, 1.0},
        }};

        std::array<T, n> h_ys {{
            0.5,
            0.5,
            0.25,
            0.75,
            0.5,
            0.5,
            0.5,
            0.25,
        }};

        std::array<split<T>, n> h_res{};
        split<T> *d_res = (split<T> *)d_res_;
        I *d_xs = (I *)d_xs_;
        T *d_ys = (T *)d_ys_;
        int n_result_bytes = n * sizeof(*d_res);
        std::array<split<T>, n> h_ref {{
            {empty, empty},
            {{entire.lb,0.0}, {0.0,entire.ub}},
            {{entire.lb,-0x1.fffffffffffffp+1023}, {-0x1.fffffffffffffp+1023,entire.ub}},
            {{entire.lb,0x1.fffffffffffffp+1023}, {0x1.fffffffffffffp+1023,entire.ub}},
            {{0.0,1.0}, {1.0,2.0}},
            {{1.0,1.0}, empty},
            {{0.0,0.5}, {0.5,1.0}},
            {{0.0,0.25}, {0.25,1.0}},
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_ys, h_ys.data(), n_bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_res, h_res.data(), n_result_bytes, hipMemcpyHostToDevice));
        test_bisect<<<numBlocks, blockSize>>>(n, d_xs, d_ys, d_res);
        CUDA_CHECK(hipMemcpy(h_res.data(), d_res, n_result_bytes, hipMemcpyDeviceToHost));
        int max_ulp_diff = 0;
        check_all_equal<split<T>, n>(h_res, h_ref, max_ulp_diff, std::source_location::current(), h_xs, h_ys);
    };

    CUDA_CHECK(hipFree(d_xs_));
    CUDA_CHECK(hipFree(d_ys_));
    CUDA_CHECK(hipFree(d_zs_));
    CUDA_CHECK(hipFree(d_res_));
}
