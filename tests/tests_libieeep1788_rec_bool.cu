#include "hip/hip_runtime.h"

#include <cuinterval/cuinterval.h>

#include "tests.h"
#include "test_ops.cuh"

#include <stdio.h>

template<typename T>
void tests_libieeep1788_rec_bool() {
    using namespace boost::ut;

    using I = interval<T>;
    using B = bool;

    I empty         = ::empty<T>();
    I entire        = ::entire<T>();
    T infinity = std::numeric_limits<T>::infinity();
    T NaN = ::nan("");

    const int n = 15; // count of largest test array
    const int n_bytes   = n * sizeof(I);
    const int blockSize = 256;
    [[maybe_unused]] const int numBlocks = (n + blockSize - 1) / blockSize;

    I *d_xs, *d_ys, *d_zs, *d_res_;

    CUDA_CHECK(hipMalloc(&d_xs, n_bytes));
    CUDA_CHECK(hipMalloc(&d_ys, n_bytes));
    CUDA_CHECK(hipMalloc(&d_zs, n_bytes));
    CUDA_CHECK(hipMalloc(&d_res_, n_bytes));

    "minimal_is_common_interval_isCommonInterval"_test = [&] {
        constexpr int n = 12;
        std::array<I, n> h_xs {{
            {-0.0,-0.0},
            {-0.0,0.0},
            {-0x1.FFFFFFFFFFFFFp1023,0x1.FFFFFFFFFFFFFp1023},
            {-27.0,-27.0},
            {-27.0,0.0},
            {-infinity,0.0},
            {0.0,-0.0},
            {0.0,0.0},
            {0.0,infinity},
            {5.0,12.4},
            empty,
            entire,
        }};

        std::array<B, n> h_res{};
        B *d_res = (B *)d_res_;
        int n_result_bytes = n * sizeof(B);
        std::array<B, n> h_ref {{
            true,
            true,
            true,
            true,
            true,
            false,
            true,
            true,
            false,
            true,
            false,
            false,
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_res, h_res.data(), n_result_bytes, hipMemcpyHostToDevice));
        test_isCommonInterval<<<numBlocks, blockSize>>>(n, d_xs, d_res);
        CUDA_CHECK(hipMemcpy(h_res.data(), d_res, n_result_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<B, n>(h_res, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("x = [%a, %a]\n", h_xs[fail_id].lb, h_xs[fail_id].ub);
        }
    };

    "minimal_is_singleton_isSingleton"_test = [&] {
        constexpr int n = 15;
        std::array<I, n> h_xs {{
            {-0.0,-0.0},
            {-0.0,0.0},
            {-1.0,-0.5},
            {-1.0,0.0},
            {-1.0,infinity},
            {-2.0,-2.0},
            {-27.0,-27.0},
            {-infinity,-0x1.FFFFFFFFFFFFFp1023},
            {0.0,-0.0},
            {0.0,0.0},
            {1.0,2.0},
            {12.0,12.0},
            {17.1,17.1},
            empty,
            entire,
        }};

        std::array<B, n> h_res{};
        B *d_res = (B *)d_res_;
        int n_result_bytes = n * sizeof(B);
        std::array<B, n> h_ref {{
            true,
            true,
            false,
            false,
            false,
            true,
            true,
            false,
            true,
            true,
            false,
            true,
            true,
            false,
            false,
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_res, h_res.data(), n_result_bytes, hipMemcpyHostToDevice));
        test_isSingleton<<<numBlocks, blockSize>>>(n, d_xs, d_res);
        CUDA_CHECK(hipMemcpy(h_res.data(), d_res, n_result_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<B, n>(h_res, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("x = [%a, %a]\n", h_xs[fail_id].lb, h_xs[fail_id].ub);
        }
    };


    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_ys));
    CUDA_CHECK(hipFree(d_zs));
    CUDA_CHECK(hipFree(d_res_));
}
