
#include <cuinterval/cuinterval.h>

#include "tests.h"
#include "test_ops.cuh"

#include <stdio.h>

template<typename T>
void tests_c_xsc() {
    using namespace boost::ut;

    using I = interval<T>;

    I empty         = ::empty<T>();
    I entire        = ::entire<T>();
    T infinity = std::numeric_limits<T>::infinity();

    const int n = 16; // count of largest test array
    const int n_bytes   = n * sizeof(I);
    const int blockSize = 256;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    interval<T> *d_xs, *d_ys, *d_zs;
    CUDA_CHECK(hipMalloc(&d_xs, n_bytes));
    CUDA_CHECK(hipMalloc(&d_ys, n_bytes));
    CUDA_CHECK(hipMalloc(&d_zs, n_bytes));

    "cxsc.intervaladdsub_add"_test = [&] {
        constexpr int n = 2;
        std::array<I, n> h_xs {{
            {10.0,20.0},
            {13.0,17.0},
        }};

        std::array<I, n> h_ys {{
            {13.0,17.0},
            {10.0,20.0},
        }};

        std::array<I, n> h_ref {{
            {23.0,37.0},
            {23.0,37.0},
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_ys, h_ys.data(), n_bytes, hipMemcpyHostToDevice));
        test_add<<<numBlocks, blockSize>>>(n, d_xs, d_ys);
        CUDA_CHECK(hipMemcpy(h_xs.data(), d_xs, n_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<I, n>(h_xs, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("y = [%a, %a]\nr = [%a, %a]\n", h_ys[fail_id].lb, h_ys[fail_id].ub, h_ref[fail_id].lb, h_ref[fail_id].ub);
        }
    };

    "cxsc.intervaladdsub_neg"_test = [&] {
        constexpr int n = 1;
        std::array<I, n> h_xs {{
            {10.0,20.0},
        }};

        std::array<I, n> h_ref {{
            {-20.0,-10.0},
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        test_neg<<<numBlocks, blockSize>>>(n, d_xs);
        CUDA_CHECK(hipMemcpy(h_xs.data(), d_xs, n_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<I, n>(h_xs, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("r = [%a, %a]\n", h_ref[fail_id].lb, h_ref[fail_id].ub);
        }
    };

    "cxsc.intervaladdsub_pos"_test = [&] {
        constexpr int n = 1;
        std::array<I, n> h_xs {{
            {10.0,20.0},
        }};

        std::array<I, n> h_ref {{
            {10.0,20.0},
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        test_pos<<<numBlocks, blockSize>>>(n, d_xs);
        CUDA_CHECK(hipMemcpy(h_xs.data(), d_xs, n_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<I, n>(h_xs, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("r = [%a, %a]\n", h_ref[fail_id].lb, h_ref[fail_id].ub);
        }
    };

    "cxsc.intervaladdsub_sub"_test = [&] {
        constexpr int n = 2;
        std::array<I, n> h_xs {{
            {10.0,20.0},
            {13.0,16.0},
        }};

        std::array<I, n> h_ys {{
            {13.0,16.0},
            {10.0,20.0},
        }};

        std::array<I, n> h_ref {{
            {-6.0,7.0},
            {-7.0,6.0},
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_ys, h_ys.data(), n_bytes, hipMemcpyHostToDevice));
        test_sub<<<numBlocks, blockSize>>>(n, d_xs, d_ys);
        CUDA_CHECK(hipMemcpy(h_xs.data(), d_xs, n_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<I, n>(h_xs, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("y = [%a, %a]\nr = [%a, %a]\n", h_ys[fail_id].lb, h_ys[fail_id].ub, h_ref[fail_id].lb, h_ref[fail_id].ub);
        }
    };

    "cxsc.intervalmuldiv_div"_test = [&] {
        constexpr int n = 16;
        std::array<I, n> h_xs {{
            {-1.0,2.0},
            {-1.0,2.0},
            {-1.0,2.0},
            {-1.0,2.0},
            {-2.0,-1.0},
            {-2.0,-1.0},
            {-2.0,-1.0},
            {-2.0,-1.0},
            {-2.0,1.0},
            {-2.0,1.0},
            {-2.0,1.0},
            {-2.0,1.0},
            {1.0,2.0},
            {1.0,2.0},
            {1.0,2.0},
            {1.0,2.0},
        }};

        std::array<I, n> h_ys {{
            {-4.0,8.0},
            {-8.0,-4.0},
            {-8.0,4.0},
            {4.0,8.0},
            {-4.0,8.0},
            {-8.0,-4.0},
            {-8.0,4.0},
            {4.0,8.0},
            {-4.0,8.0},
            {-8.0,-4.0},
            {-8.0,4.0},
            {4.0,8.0},
            {-4.0,8.0},
            {-8.0,-4.0},
            {-8.0,4.0},
            {4.0,8.0},
        }};

        std::array<I, n> h_ref {{
            entire,
            {-0.5,0.25},
            entire,
            {-0.25,0.5},
            entire,
            {0.125,0.5},
            entire,
            {-0.5,-0.125},
            entire,
            {-0.25,0.5},
            entire,
            {-0.5,0.25},
            entire,
            {-0.5,-0.125},
            entire,
            {0.125,0.5},
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_ys, h_ys.data(), n_bytes, hipMemcpyHostToDevice));
        test_div<<<numBlocks, blockSize>>>(n, d_xs, d_ys);
        CUDA_CHECK(hipMemcpy(h_xs.data(), d_xs, n_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<I, n>(h_xs, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("y = [%a, %a]\nr = [%a, %a]\n", h_ys[fail_id].lb, h_ys[fail_id].ub, h_ref[fail_id].lb, h_ref[fail_id].ub);
        }
    };

    "cxsc.intervalmuldiv_mul"_test = [&] {
        constexpr int n = 15;
        std::array<I, n> h_xs {{
            {-1.0,2.0},
            {-1.0,2.0},
            {-1.0,2.0},
            {-1.0,2.0},
            {-2.0,-1.0},
            {-2.0,-1.0},
            {-2.0,-1.0},
            {-2.0,-1.0},
            {-2.0,1.0},
            {-2.0,1.0},
            {-2.0,1.0},
            {1.0,2.0},
            {1.0,2.0},
            {1.0,2.0},
            {1.0,2.0},
        }};

        std::array<I, n> h_ys {{
            {-3.0,4.0},
            {-4.0,-3.0},
            {-4.0,3.0},
            {3.0,4.0},
            {-3.0,4.0},
            {-4.0,-3.0},
            {-4.0,3.0},
            {3.0,4.0},
            {-3.0,4.0},
            {-4.0,3.0},
            {3.0,4.0},
            {-3.0,4.0},
            {-4.0,-3.0},
            {-4.0,3.0},
            {3.0,4.0},
        }};

        std::array<I, n> h_ref {{
            {-6.0,8.0},
            {-8.0,4.0},
            {-8.0,6.0},
            {-4.0,8.0},
            {-8.0,6.0},
            {3.0,8.0},
            {-6.0,8.0},
            {-8.0,-3.0},
            {-8.0,6.0},
            {-6.0,8.0},
            {-8.0,4.0},
            {-6.0,8.0},
            {-8.0,-3.0},
            {-8.0,6.0},
            {3.0,8.0},
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_ys, h_ys.data(), n_bytes, hipMemcpyHostToDevice));
        test_mul<<<numBlocks, blockSize>>>(n, d_xs, d_ys);
        CUDA_CHECK(hipMemcpy(h_xs.data(), d_xs, n_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<I, n>(h_xs, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("y = [%a, %a]\nr = [%a, %a]\n", h_ys[fail_id].lb, h_ys[fail_id].ub, h_ref[fail_id].lb, h_ref[fail_id].ub);
        }
    };

    "cxsc.intervalstdfunc_sqr"_test = [&] {
        constexpr int n = 3;
        std::array<I, n> h_xs {{
            {-9.0,-9.0},
            {0.0,0.0},
            {11.0,11.0},
        }};

        std::array<I, n> h_ref {{
            {81.0,81.0},
            {0.0,0.0},
            {121.0,121.0},
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        test_sqr<<<numBlocks, blockSize>>>(n, d_xs);
        CUDA_CHECK(hipMemcpy(h_xs.data(), d_xs, n_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<I, n>(h_xs, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("r = [%a, %a]\n", h_ref[fail_id].lb, h_ref[fail_id].ub);
        }
    };

    "cxsc.intervalstdfunc_sqrt"_test = [&] {
        constexpr int n = 3;
        std::array<I, n> h_xs {{
            {0.0,0.0},
            {121.0,121.0},
            {81.0,81.0},
        }};

        std::array<I, n> h_ref {{
            {0.0,0.0},
            {11.0,11.0},
            {9.0,9.0},
        }};

        CUDA_CHECK(hipMemcpy(d_xs, h_xs.data(), n_bytes, hipMemcpyHostToDevice));
        test_sqrt<<<numBlocks, blockSize>>>(n, d_xs);
        CUDA_CHECK(hipMemcpy(h_xs.data(), d_xs, n_bytes, hipMemcpyDeviceToHost));
        auto failed = check_all_equal<I, n>(h_xs, h_ref);
        for (auto fail_id : failed) {
            printf("failed at case %zu:\n", fail_id);
            printf("r = [%a, %a]\n", h_ref[fail_id].lb, h_ref[fail_id].ub);
        }
    };


    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_ys));
    CUDA_CHECK(hipFree(d_zs));
}
