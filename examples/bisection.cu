
#include <cstdlib>

#include "cuinterval/examples/bisection.cuh"
#include <cuinterval/cuinterval.h>

#include "utils.h"

void example_bisection()
{
    using I = interval<double>;

    I x = { -5.0, 10.0 };

    constexpr double tolerance      = 1e-12;
    constexpr std::size_t max_depth = 512;
    std::size_t max_roots           = 16;

    std::size_t *d_max_roots;
    CUDA_CHECK(hipMalloc(&d_max_roots, sizeof(*d_max_roots)));
    CUDA_CHECK(hipMemcpy(d_max_roots, &max_roots, sizeof(*d_max_roots), hipMemcpyHostToDevice));

    I *d_roots;
    CUDA_CHECK(hipMalloc(&d_roots, max_roots * sizeof(*d_roots)));
    bisection<double, max_depth><<<1, 1>>>(x, tolerance, d_roots, d_max_roots);
    CUDA_CHECK(hipMemcpy(&max_roots, d_max_roots, sizeof(*d_max_roots), hipMemcpyDeviceToHost));
    printf("We found %zu roots.\n", max_roots);

    I *h_roots = (I *)std::malloc(max_roots * sizeof(*h_roots));
    CUDA_CHECK(hipMemcpy(h_roots, d_roots, max_roots * sizeof(I), hipMemcpyDeviceToHost));
    for (std::size_t i = 0; i < max_roots; i++) {
        printf("Root %zu in [%.15f, %.15f]\n", i, h_roots[i].lb, h_roots[i].ub);


        printf("diff is: %.15f\n", h_roots[i].ub - h_roots[i].lb);
        printf("tolerance is: %.15f\n", tolerance);
    }


    std::free(h_roots);
    CUDA_CHECK(hipFree(d_roots));
    CUDA_CHECK(hipFree(d_max_roots));
}

int main()
{
    example_bisection();
    return 0;
}
