#include "hip/hip_runtime.h"

#include <cstdlib>

#include "cuinterval/examples/bisection.cuh"
#include <cuinterval/cuinterval.h>

#include "utils.h"

using cu::interval;

template<typename I>
__device__ I f(I x)
{
//     // return exp(I { -3.0, -3.0 } * x) - sin(x) * sin(x) * sin(x);
//     // return I{1.0, 1.0};
//     // return x*sqr(x) - (I{2.0, 2.0} * sqr(x)) + x;
//     // return sqr(sin(x)) - (I{1.0, 1.0} - cos(I{2.0, 2.0} * x)) / I{2.0, 2.0};
    return pow(x, 3) - pow(x, 2) - 17.0 * x - 15.0;
};

__device__ fn_t d_f = f<interval<double>>;

void example_bisection()
{
    using I = interval<double>;

    I x = { -5.0, 10.0 };

    constexpr double tolerance      = 1e-12;
    constexpr std::size_t max_depth = 512;
    std::size_t max_roots           = 16;

    std::size_t *d_max_roots;
    CUDA_CHECK(hipMalloc(&d_max_roots, sizeof(*d_max_roots)));
    CUDA_CHECK(hipMemcpy(d_max_roots, &max_roots, sizeof(*d_max_roots), hipMemcpyHostToDevice));

    I *d_roots;
    CUDA_CHECK(hipMalloc(&d_roots, max_roots * sizeof(*d_roots)));

    fn_t h_f;
    hipMemcpyFromSymbol(&h_f, HIP_SYMBOL(d_f), sizeof(h_f));

    bisection<double, max_depth><<<1, 1>>>(h_f, x, tolerance, d_roots, d_max_roots);
    CUDA_CHECK(hipMemcpy(&max_roots, d_max_roots, sizeof(*d_max_roots), hipMemcpyDeviceToHost));

    I *h_roots = (I *)std::malloc(max_roots * sizeof(*h_roots));
    CUDA_CHECK(hipMemcpy(h_roots, d_roots, max_roots * sizeof(I), hipMemcpyDeviceToHost));
    for (std::size_t i = 0; i < max_roots; i++) {
        printf("Root %zu in [%.15f, %.15f]\n", i, h_roots[i].lb, h_roots[i].ub);
    }

    std::free(h_roots);
    CUDA_CHECK(hipFree(d_roots));
    CUDA_CHECK(hipFree(d_max_roots));
}

int main()
{
    example_bisection();
    return 0;
}
