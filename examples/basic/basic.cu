#include "hip/hip_runtime.h"
#include <cuinterval/cuinterval.h>

#include <hip/hip_runtime.h>

#define CUDA_CHECK(x)                                                                \
    do {                                                                             \
        hipError_t err = x;                                                         \
        if (err != hipSuccess) {                                                    \
            fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, \
                    __FILE__, __LINE__, hipGetErrorString(err),                     \
                    hipGetErrorName(err), err);                                     \
            abort();                                                                 \
        }                                                                            \
    } while (0)

__device__ auto f(auto x, auto y)
{
    return pow(x - 1.0, 3) - sqr(x) + 4.0;
}

__global__ void kernel(auto *xs, auto *ys, auto *res, std::integral auto n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        res[i] = f(xs[i], ys[i]);
    }
}

int main()
{
    constexpr int n = 256;
    using T         = cu::interval<double>;
    T xs[n], ys[n], res[n];

    // generate dummy data
    for (int i = 0; i < n; i++) {
        double v = i;
        xs[i]    = { { .lb = 0.0, .ub = v } };
        ys[i]    = { 0.0, v };
    }

    T *d_xs, *d_ys, *d_res;
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_ys, n * sizeof(*ys)));
    CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));

    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ys, ys, n * sizeof(*ys), hipMemcpyHostToDevice));

    kernel<<<n, 1>>>(d_xs, d_ys, d_res, n);

    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    auto r = res[3];
    printf("f([0,3], [0,3]) = [%g, %g]\n", r.lb, r.ub);

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_ys));
    CUDA_CHECK(hipFree(d_res));

    return 0;
}
