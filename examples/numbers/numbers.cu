#include "hip/hip_runtime.h"
#include <cuinterval/cuinterval.h>

#include <hip/hip_runtime.h>

#define CUDA_CHECK(x)                                                                \
    do {                                                                             \
        hipError_t err = x;                                                         \
        if (err != hipSuccess) {                                                    \
            fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, \
                    __FILE__, __LINE__, hipGetErrorString(err),                     \
                    hipGetErrorName(err), err);                                     \
            abort();                                                                 \
        }                                                                            \
    } while (0)

template <typename T>
__device__ T area_of_circle(T r) {
    return std::numbers::pi_v<T> * r * r;
}

__global__ void kernel(auto *xs, auto *res, std::integral auto n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        res[i] = area_of_circle(xs[i]);
    }
}

int main()
{
    constexpr int n = 16;
    using T         = cu::interval<double>;
    T xs[n], res[n];

    // generate dummy data
    for (int i = 0; i < n; i++) {
        double v = i;
        xs[i]    = { v };
    }

    T *d_xs, *d_res;
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));

    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));

    kernel<<<n, 1>>>(d_xs, d_res, n);

    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    for (int i = 0; i < n; i++) {
        auto r = res[i];
        printf("area_of_circle(%g) = [%.15f, %.15f]\n", xs[i].lb, r.lb, r.ub);
    }

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_res));

    return 0;
}
